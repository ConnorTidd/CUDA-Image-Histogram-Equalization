#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
 
#include "config.h"

#define TIMER_CREATE(t)               \
  hipEvent_t t##_start, t##_end;     \
  hipEventCreate(&t##_start);        \
  hipEventCreate(&t##_end);               
 
#define TIMER_START(t)                \
  hipEventRecord(t##_start);         \
  hipEventSynchronize(t##_start);    \
 
#define TIMER_END(t)                             \
  hipEventRecord(t##_end);                      \
  hipEventSynchronize(t##_end);                 \
  hipEventElapsedTime(&t, t##_start, t##_end);  \
  hipEventDestroy(t##_start);                   \
  hipEventDestroy(t##_end);     

/*******************************************************/
/*                 Cuda Error Function                 */
/*******************************************************/
inline hipError_t checkCuda(hipError_t result) {
	#if defined(DEBUG) || defined(_DEBUG)
		if (result != hipSuccess) {
			fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
			exit(-1);
		}
	#endif
		return result;
}
      

__global__ void createHistogram(unsigned char *input, 
    int *histogram,
    unsigned int width,
    unsigned int height) {

    int x = blockIdx.x*TILE_SIZE+threadIdx.x;
    int y = blockIdx.y*TILE_SIZE+threadIdx.y;

    int location = 	y*TILE_SIZE*gridDim.x+x;

    // int pixels = width * height;
    int val = input[location];
    atomicAdd(&(histogram[val]), 1);

}
// Add GPU kernel and functions
// HERE!!!
__global__ void kernel(unsigned char *input, 
                       unsigned char *output,
                       unsigned int *normed_hist,
                       unsigned int width,
                       unsigned int height){

    int x = blockIdx.x*TILE_SIZE+threadIdx.x;
    int y = blockIdx.y*TILE_SIZE+threadIdx.y;

    int location = 	y*TILE_SIZE*gridDim.x+x;
    // data is the copy
    int pixels = width*height;
    int val = input[location];
    unsigned char dumbbutt = (normed_hist[val])*255.0/pixels;
    output[location] = dumbbutt;
}

__global__ void warmup(unsigned char *input, 
                       unsigned char *output){

	int x = blockIdx.x*TILE_SIZE+threadIdx.x;
	int y = blockIdx.y*TILE_SIZE+threadIdx.y;
	  
	int location = 	y*(gridDim.x*TILE_SIZE)+x;
	
    output[location] = 0;

}

// NOTE: The data passed on is already padded
void gpu_function(unsigned char *data,  
                  unsigned int height, 
                  unsigned int width){
    
    unsigned char *input_gpu;
    unsigned char *output_gpu;
    unsigned int *normed_hist;
    int *histogram_d;

	int gridXSize = 1 + (( width - 1) / TILE_SIZE);
	int gridYSize = 1 + ((height - 1) / TILE_SIZE);
	
	int XSize = gridXSize*TILE_SIZE;
	int YSize = gridYSize*TILE_SIZE;
	
	int size = XSize*YSize;
	
	// Allocate arrays in GPU memory
	checkCuda(hipMalloc((void**)&input_gpu   , size*sizeof(unsigned char)));
    checkCuda(hipMalloc((void**)&normed_hist , 256*sizeof(unsigned int)));
    checkCuda(hipMalloc((void**)&output_gpu  , size*sizeof(unsigned char)));
    checkCuda(hipMalloc((void**)&histogram_d , 256 * sizeof(int)));
	
    checkCuda(hipMemset(output_gpu , 0 , size*sizeof(unsigned char)));
    checkCuda(hipMemset(normed_hist , 0 , 256*sizeof(unsigned int)));
    checkCuda(hipMemset(histogram_d , 0 , 256*sizeof(int)));
    // Copy data to GPU
    

    // Execute algorithm

    dim3 dimGrid(gridXSize, gridYSize);
    dim3 dimBlock(TILE_SIZE, TILE_SIZE);

	

     
    // int pixels = height*width;
    printf("\n");



    // call kernel
    // Add more kernels and functions as needed here
    createHistogram<<<dimGrid, dimBlock>>>(input_gpu,
        histogram_d,
        width,
        height);

    int *histogram_host = new int [256];
    checkCuda(hipMemcpy(histogram_host, histogram_d, 256*sizeof(int), hipMemcpyDeviceToHost));
    

    unsigned int* cumhistogram = new unsigned int[256];
    cumhistogram[0] = histogram_host[0];
    
    hipHostMalloc(&cumhistogram, sizeof(unsigned int)*256);
    for(int i = 1; i < 256; i = i + 1) {
        cumhistogram[i] = histogram_host[i] + cumhistogram[i-1];
    }
    
    checkCuda(hipMemcpy(input_gpu, 
        data, 
        size*sizeof(unsigned char), 
        hipMemcpyHostToDevice));

    checkCuda(hipMemcpy(normed_hist, 
        cumhistogram, 
        256*sizeof(unsigned int), 
        hipMemcpyHostToDevice));


	checkCuda(hipDeviceSynchronize());

    // Kernel Call
	#ifdef CUDA_TIMING
         float Ktime;
        TIMER_CREATE(Ktime);
        TIMER_START(Ktime);
    #endif
    
    // Add more kernels and functions as needed here
    kernel<<<dimGrid, dimBlock>>>(input_gpu,
                                  output_gpu,
                                  normed_hist,
                                  width,
                                  height);
    
    // From here on, no need to change anything
    checkCuda(hipPeekAtLastError());                                     
    checkCuda(hipDeviceSynchronize());
    
    #ifdef CUDA_TIMING 
        TIMER_END(Ktime);
        printf("Kernel Execution Time: %f ms\n", Ktime);
    #endif
    
    // Retrieve results from the GPU

    
    checkCuda(hipMemcpy(data, 
        output_gpu, 
        size*sizeof(unsigned char), 
        hipMemcpyDeviceToHost));
        
    printf("\ntotal is:%d\n", height*width);
    // Free resources and end the program
    //free(mapf);
    //free(data);
	checkCuda(hipFree(output_gpu));
    checkCuda(hipFree(input_gpu));
    checkCuda(hipFree(normed_hist));
    checkCuda(hipFree(histogram_d));

    // checkCuda(hipFree(cumhistogram));
}












































void gpu_warmup(unsigned char *data, 
                unsigned int height, 
                unsigned int width){
    
    unsigned char *input_gpu;
    unsigned char *output_gpu;
     
	int gridXSize = 1 + (( width - 1) / TILE_SIZE);
	int gridYSize = 1 + ((height - 1) / TILE_SIZE);
	
	int XSize = gridXSize*TILE_SIZE;
	int YSize = gridYSize*TILE_SIZE;
	
	// Both are the same size (CPU/GPU).
	int size = XSize*YSize;
	
	// Allocate arrays in GPU memory
	checkCuda(hipMalloc((void**)&input_gpu   , size*sizeof(unsigned char)));
	checkCuda(hipMalloc((void**)&output_gpu  , size*sizeof(unsigned char)));
	
    checkCuda(hipMemset(output_gpu , 0 , size*sizeof(unsigned char)));
            
    // Copy data to GPU
    checkCuda(hipMemcpy(input_gpu, 
        data, 
        size*sizeof(char), 
        hipMemcpyHostToDevice));

	checkCuda(hipDeviceSynchronize());
        
    // Execute algorithm
        
	dim3 dimGrid(gridXSize, gridYSize);
    dim3 dimBlock(TILE_SIZE, TILE_SIZE);
    
    warmup<<<dimGrid, dimBlock>>>(input_gpu, 
                                  output_gpu);
                                         
    checkCuda(hipDeviceSynchronize());
        
	// Retrieve results from the GPU
	checkCuda(hipMemcpy(data, 
			output_gpu, 
			size*sizeof(unsigned char), 
			hipMemcpyDeviceToHost));
                        
    // Free resources and end the program
	checkCuda(hipFree(output_gpu));
    checkCuda(hipFree(input_gpu));

}

